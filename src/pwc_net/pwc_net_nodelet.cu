#include "hip/hip_runtime.h"
#include <pluginlib/class_list_macros.h>

#include "pwc_net_nodelet.h"

PLUGINLIB_EXPORT_CLASS(pwc_net::PWCNetNodelet, nodelet::Nodelet)

#include <cv_bridge/cv_bridge.h>
#include <ros/package.h>
#include <ros/time.h>

#include <cmath>
#include <cstdlib>
#include <fstream>
#include <map>
#include <vector>

namespace pwc_net{

void PWCNetNodelet::onInit() {
  ros::NodeHandle& node_handle = getNodeHandle();
  ros::NodeHandle& private_node_handle = getPrivateNodeHandle();

  scale_ratio_ = private_node_handle.param("scale_ratio", 1.0);

  // Initialize network if image_width and image_height params are available
  int image_width, image_height;
  bool width_param = private_node_handle.getParam("image_width", image_width);
  bool height_param = private_node_handle.getParam("image_height", image_height);
  if (width_param && height_param)
    initializeNetwork(image_width, image_height);

  std::string image_topic = node_handle.resolveName("image");
  image_transport_.reset(new image_transport::ImageTransport(node_handle));
  image_subscriber_ = image_transport_->subscribe(image_topic, 1, &PWCNetNodelet::imageCallback, this);
  
  flow_publisher_ = private_node_handle.advertise<optical_flow_msgs::DenseOpticalFlow>("optical_flow", 1);

  flow_service_server_ = private_node_handle.advertiseService("calculate_dense_optical_flow", &PWCNetNodelet::serviceCallback, this);
}

std::string PWCNetNodelet::generateTemporaryModelFile(const std::string &package_path) {
  std::string model_file_template = package_path + "/model/pwc_net_test.prototxt";
  NODELET_INFO_STREAM("Loading template of model file: " << model_file_template);

  std::ifstream template_ifstream(model_file_template);
  if (!template_ifstream.is_open()) {
    NODELET_FATAL_STREAM("Cannot open template file of model: " << model_file_template);
    ros::shutdown();
    exit(EXIT_FAILURE);
  }

  std::string temporary_model_file = package_path + "/model/tmp/tmp_model_file.prototxt";
  NODELET_INFO_STREAM("Generate temporary model file: " << temporary_model_file);

  std::ofstream temporary_ofstream(temporary_model_file);
  if (!temporary_ofstream.is_open()) {
    NODELET_FATAL_STREAM("Cannot open temporary model file: " << temporary_model_file);
    ros::shutdown();
    exit(EXIT_FAILURE);
  }

  std::map<std::string, std::string> replacement_map;
  replacement_map.emplace("$ADAPTED_WIDTH", std::to_string(adapted_width_));
  replacement_map.emplace("$ADAPTED_HEIGHT", std::to_string(adapted_height_));
  replacement_map.emplace("$TARGET_WIDTH", std::to_string(target_width_));
  replacement_map.emplace("$TARGET_HEIGHT", std::to_string(target_height_));
  replacement_map.emplace("$SCALE_WIDTH", std::to_string(1.0 * target_width_ / adapted_width_));
  replacement_map.emplace("$SCALE_HEIGHT", std::to_string(1.0 * target_height_ / adapted_height_));

  for (std::string line; std::getline(template_ifstream, line);) {
    for (auto iterator = replacement_map.begin(); iterator != replacement_map.end(); iterator++) {
      size_t replace_start;
      std::string replaced_text = iterator->first;
      std::string new_text = iterator->second;

      replace_start = line.find(replaced_text);
      if (replace_start != std::string::npos)
        line.replace(replace_start, replaced_text.length(), new_text);
    }

    line += "\n";
    temporary_ofstream.write(line.c_str(), static_cast<long>(line.length()));
  }
  template_ifstream.close();
  temporary_ofstream.close();

  return temporary_model_file;
}

void PWCNetNodelet::imageCallback(const sensor_msgs::ImageConstPtr& image_msg) {
  ros::WallTime process_start = ros::WallTime::now();

  cv::Mat current_image;
  try {
    current_image = cv_bridge::toCvCopy(image_msg, "bgr8")->image;
  }
  catch(const cv_bridge::Exception& exception) {
    NODELET_ERROR_STREAM(exception.what());
    return;
  }

  if (!net_)
    initializeNetwork(current_image.cols, current_image.rows);

  if (current_image.cols != target_width_ || current_image.rows != target_height_) {
    NODELET_ERROR_STREAM("Size of current image is not same to first input image which is used to initialize network.\n" << 
      "first: " << target_width_ << "x" << target_height_ << "\n" <<
      "current: " << current_image.cols << "x" << current_image.rows);
    return;
  }

  // Convert image to float for input layer
  current_image.convertTo(current_image, CV_32FC3);

  if (!previous_image_.empty()) {
    setImagesToInputLayer(previous_image_, current_image);

    ros::WallTime inference_start = ros::WallTime::now();
    net_->Forward();
    ros::WallDuration inference_time = ros::WallTime::now() - inference_start;

    publishOpticalFlow(image_msg->header);
    ros::WallDuration process_time = ros::WallTime::now() - process_start;

    NODELET_INFO_STREAM("Total process time: " << process_time.toSec() << " [s] (inference time: " << inference_time.toSec() << " [s])");
  }
  
  current_image.copyTo(previous_image_);
  previous_stamp_ = image_msg->header.stamp;
}

void PWCNetNodelet::initializeNetwork(int image_width, int image_height) {
  NODELET_INFO_STREAM("Start network initialization\n"
    << "input image size: " << image_width << "x" << image_height);

  if (image_width <= 0 || image_height <= 0)
  {
    NODELET_FATAL_STREAM("Invalid size is specified to network initialization!\n" 
      << "Specified value: " << image_width << "x" << image_height);
      ros::shutdown();
      std::exit(EXIT_FAILURE);
  }

  target_width_ = image_width;
  target_height_ = image_height;
  adapted_width_ = static_cast<int>(std::ceil(target_width_ / RESOLUTION_DIVISOR_ * scale_ratio_) * RESOLUTION_DIVISOR_);
  adapted_height_ = static_cast<int>(std::ceil(target_height_ / RESOLUTION_DIVISOR_ * scale_ratio_) * RESOLUTION_DIVISOR_);

  std::string package_path = ros::package::getPath(PACKAGE_NAME_);
  if (package_path.empty()) {
    NODELET_FATAL_STREAM("Package not found: " << PACKAGE_NAME_);
    ros::shutdown();
    std::exit(EXIT_FAILURE);
  }

  std::string temporary_model_file = generateTemporaryModelFile(package_path);

  NODELET_INFO("Loading temporary model file");
  net_.reset(new caffe::Net<d_type_>(temporary_model_file, caffe::TEST));

  std::string trained_file = package_path + "/model/pwc_net.caffemodel";
  NODELET_INFO_STREAM("Loading trained file: " << trained_file);
  net_->CopyTrainedLayersFrom(trained_file);

  NODELET_INFO_STREAM("Network initialization is finished");
}

void PWCNetNodelet::outputLayerToFlowMsg(const std::string& frame_id, const ros::Time& newer_stamp, const ros::Time& older_stamp, optical_flow_msgs::DenseOpticalFlow* flow_msg)
{
  const boost::shared_ptr<caffe::Blob<d_type_>> output_blob = net_->blob_by_name(OUTPUT_BLOB_);

  flow_msg->header.frame_id = frame_id;
  flow_msg->header.stamp = newer_stamp;
  flow_msg->previous_stamp = older_stamp;

  flow_msg->width = output_blob->shape(3);
  flow_msg->height = output_blob->shape(2);

  size_t flow_num = flow_msg->width * flow_msg->height;
  flow_msg->invalid_map.resize(flow_num, false);
  flow_msg->flow_field.resize(flow_num);

  const float *flow_x = output_blob->cpu_data();
  const float *flow_y = flow_x + flow_num;

  for (int i = 0; i < flow_num; i++) {
    optical_flow_msgs::PixelDisplacement& flow_at_point = flow_msg->flow_field[i];
    flow_at_point.x = flow_x[i];
    flow_at_point.y = flow_y[i];
  }
}

void PWCNetNodelet::publishOpticalFlow(const std_msgs::Header& current_image_header) {
  const boost::shared_ptr<caffe::Blob<d_type_>> output_blob = net_->blob_by_name(OUTPUT_BLOB_);

  optical_flow_msgs::DenseOpticalFlow flow_msg;
  outputLayerToFlowMsg(current_image_header.frame_id, current_image_header.stamp, previous_stamp_, &flow_msg);

  flow_publisher_.publish(flow_msg);
}

bool PWCNetNodelet::serviceCallback(optical_flow_srvs::CalculateDenseOpticalFlow::Request& request, optical_flow_srvs::CalculateDenseOpticalFlow::Response& response)
{
  ros::WallTime process_start = ros::WallTime::now();
  NODELET_INFO("CalculateDenseOpticalFlow service is called.");

  cv::Mat older_image, newer_image;
  try {
    older_image = cv_bridge::toCvCopy(request.older_image, "bgr8")->image;
    newer_image = cv_bridge::toCvCopy(request.newer_image, "bgr8")->image;
  }
  catch(const cv_bridge::Exception& exception) {
    NODELET_ERROR_STREAM(exception.what());
    return false;
  }

  if (older_image.cols != newer_image.cols || older_image.rows != newer_image.rows)
  {
    NODELET_ERROR_STREAM("Two images in request is not same size.\n"
      << "older_image: " << older_image.cols << "x" << older_image.rows << "\n"
      << "newer_image: " << newer_image.cols << "x" << newer_image.rows);
    return false;
  }

  if (!net_ || newer_image.cols != target_width_ || newer_image.rows != target_height_)
    initializeNetwork(newer_image.cols, newer_image.rows);

  // Convert image to float for input layer
  older_image.convertTo(older_image, CV_32FC3);
  newer_image.convertTo(newer_image, CV_32FC3);

  setImagesToInputLayer(older_image, newer_image);

  ros::WallTime inference_start = ros::WallTime::now();
  net_->Forward();
  ros::WallDuration inference_time = ros::WallTime::now() - inference_start;

  std::string& frame_id = request.newer_image.header.frame_id;
  ros::Time& newer_stamp = request.newer_image.header.stamp;
  ros::Time& older_stamp = request.older_image.header.stamp;
  outputLayerToFlowMsg(frame_id, newer_stamp, older_stamp, &response.optical_flow);

  ros::WallDuration process_time = ros::WallTime::now() - process_start;
  NODELET_INFO_STREAM("Service process time: " << process_time.toSec() << " [s] (inference time: " << inference_time.toSec() << " [s])");

  return true;
}

void PWCNetNodelet::setImagesToInputLayer(const cv::Mat& older_image, const cv::Mat& newer_image) {
  std::vector<cv::Mat> channels;
  size_t channel_size = older_image.cols * older_image.rows;

  // Set older image
  cv::split(older_image, channels); // Split to BGR channels
  d_type_ *input_layer_blob = net_->blob_by_name(INPUT_BLOB_OLDER_)->mutable_cpu_data();
  // Store each channels to blob
  for (int i = 0; i < 3; i++)
    memcpy(input_layer_blob + (channel_size * i), channels[i].ptr<d_type_>(), channel_size * sizeof(float));

  // Set current image
  cv::split(newer_image, channels);
  input_layer_blob = net_->blob_by_name(INPUT_BLOB_NEWER_)->mutable_cpu_data();
  for (int i = 0; i < 3; i++)
    memcpy(input_layer_blob + (channel_size * i), channels[i].ptr<d_type_>(), channel_size * sizeof(float));

  caffe::Caffe::set_mode(caffe::Caffe::GPU);
}

}
