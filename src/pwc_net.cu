#include "hip/hip_runtime.h"
#include "pwc_net/pwc_net.h"

#include <cv_bridge/cv_bridge.h>
#include <ros/console.h>
#include <ros/package.h>

#include <caffe/caffe.hpp>
#include <caffe/layers/input_layer.hpp>

#include <cmath>
#include <fstream>
#include <map>

namespace pwc_net{

// Put here to include pwc_net.h from non-CUDA project
std::shared_ptr<caffe::Net<float>> net_;

std::string PwcNet::generateTemporaryModelFile(const std::string &package_path) {
  std::string model_file_template = package_path + "/model/pwc_net_test.prototxt";
  ROS_INFO_STREAM_NAMED("libpwc_net", "Loading template of model file: " << model_file_template);

  std::ifstream template_ifstream(model_file_template);
  if (!template_ifstream.is_open()) {
    ROS_FATAL_STREAM_NAMED("libpwc_net", "Cannot open template file of model: " << model_file_template);
    ros::shutdown();
    exit(EXIT_FAILURE);
  }

  std::string temporary_model_file = package_path + "/model/tmp_model_file.prototxt";
  ROS_INFO_STREAM_NAMED("libpwc_net", "Generate temporary model file: " << temporary_model_file);

  std::ofstream temporary_ofstream(temporary_model_file);
  if (!temporary_ofstream.is_open()) {
    ROS_FATAL_STREAM_NAMED("libpwc_net", "Cannot open temporary model file: " << temporary_model_file);
    ros::shutdown();
    exit(EXIT_FAILURE);
  }

  std::map<std::string, std::string> replacement_map;
  replacement_map.emplace("$ADAPTED_WIDTH", std::to_string(adapted_width_));
  replacement_map.emplace("$ADAPTED_HEIGHT", std::to_string(adapted_height_));
  replacement_map.emplace("$TARGET_WIDTH", std::to_string(target_width_));
  replacement_map.emplace("$TARGET_HEIGHT", std::to_string(target_height_));
  replacement_map.emplace("$SCALE_WIDTH", std::to_string(1.0 * target_width_ / adapted_width_));
  replacement_map.emplace("$SCALE_HEIGHT", std::to_string(1.0 * target_height_ / adapted_height_));

  for (std::string line; std::getline(template_ifstream, line);) {
    for (auto iterator = replacement_map.begin(); iterator != replacement_map.end(); iterator++) {
      size_t replace_start;
      std::string replaced_text = iterator->first;
      std::string new_text = iterator->second;

      replace_start = line.find(replaced_text);
      if (replace_start != std::string::npos)
        line.replace(replace_start, replaced_text.length(), new_text);
    }

    line += "\n";
    temporary_ofstream.write(line.c_str(), static_cast<long>(line.length()));
  }
  template_ifstream.close();
  temporary_ofstream.close();

  return temporary_model_file;
}

bool PwcNet::estimateOpticalFlow
(
  const sensor_msgs::Image& source_image_msg, 
  const sensor_msgs::Image& dist_image_msg,
  cv::Mat& optical_flow
) 
{
  // Are input images same size?
  if (source_image_msg.width != dist_image_msg.width
    || source_image_msg.height != dist_image_msg.height)
  {
    ROS_ERROR_STREAM_NAMED("libpwc_net",
      "Input images aren't same size!\n" <<
      "source: " << source_image_msg.width << "x" << source_image_msg.height << "\n" <<
      "dist: " << dist_image_msg.width << "x" << dist_image_msg.height
    );

    return false;
  }

  // Initialize network if not
  if (!net_)
    initializeNetwork(dist_image_msg.width, dist_image_msg.height);
  else if (dist_image_msg.width != target_width_ || dist_image_msg.height != target_height_)
  {
    ROS_INFO_STREAM_NAMED("libpwc_net", 
      "Size of input image is not same to first input image which is used to initialize network.\n" << 
      "Reinitialize network for new size\n" << 
      "old: " << target_width_ << "x" << target_height_ << "\n" <<
      "new: " << dist_image_msg.width << "x" << dist_image_msg.height
    );
    initializeNetwork(dist_image_msg.width, dist_image_msg.height);
  }

  // Convert msg to cv::Mat
  cv::Mat dist_image;
  cv::Mat source_image;
  try 
  {
    dist_image = cv_bridge::toCvCopy(dist_image_msg, "bgr8")->image;
    source_image = cv_bridge::toCvCopy(source_image_msg, "bgr8")->image;
  }
  catch(const cv_bridge::Exception& exception) 
  {
    ROS_ERROR_STREAM_NAMED("libpwc_net", exception.what());
    return false;
  }

  // Convert cv::Mat to float and set to input layer
  source_image.convertTo(source_image, CV_32FC3);
  dist_image.convertTo(dist_image, CV_32FC3);
  setImagesToInputLayer(source_image, dist_image);

  net_->Forward();

  outputLayerToCvMat(optical_flow);

  return true;
}

void PwcNet::initializeNetwork(int image_width, int image_height) {
  ROS_INFO_STREAM_NAMED("libpwc_net", "Start network initialization\n"
    << "input image size: " << image_width << "x" << image_height);

  if (image_width <= 0 || image_height <= 0)
  {
    ROS_FATAL_STREAM_NAMED("libpwc_net", "Invalid size is specified to network initialization!\n" 
      << "Specified value: " << image_width << "x" << image_height);
      ros::shutdown();
      std::exit(EXIT_FAILURE);
  }

  target_width_ = image_width;
  target_height_ = image_height;
  adapted_width_ = static_cast<int>(std::ceil(target_width_ / RESOLUTION_DIVISOR_) * RESOLUTION_DIVISOR_);
  adapted_height_ = static_cast<int>(std::ceil(target_height_ / RESOLUTION_DIVISOR_) * RESOLUTION_DIVISOR_);

  std::string package_path = ros::package::getPath(PACKAGE_NAME_);
  if (package_path.empty()) {
    ROS_FATAL_STREAM_NAMED("libpwc_net", "Package not found: " << PACKAGE_NAME_);
    ros::shutdown();
    std::exit(EXIT_FAILURE);
  }

  std::string temporary_model_file = generateTemporaryModelFile(package_path);

  ROS_INFO_NAMED("libpwc_net", "Loading temporary model file");
  net_.reset(new caffe::Net<float>(temporary_model_file, caffe::TEST));

  std::string trained_file = package_path + "/model/pwc_net.caffemodel";
  ROS_INFO_STREAM_NAMED("libpwc_net", "Loading trained file: " << trained_file);
  net_->CopyTrainedLayersFrom(trained_file);

  ROS_INFO_STREAM_NAMED("libpwc_net", "Network initialization is finished");
}

void PwcNet::outputLayerToCvMat(cv::Mat& optical_flow)
{
  const boost::shared_ptr<caffe::Blob<float>> output_blob = net_->blob_by_name(OUTPUT_BLOB_);

  cv::Mat channels[2];

  int height = output_blob->shape(2);
  int width = output_blob->shape(3);

  channels[0].create(cv::Size(width, height), CV_32FC1);
  channels[1].create(cv::Size(width, height), CV_32FC1);

  int total_pixel = height * width;
  const float* x_channel = output_blob->cpu_data();
  const float* y_channel = x_channel + total_pixel;
  size_t channel_size = sizeof(float) * total_pixel;
  std::memcpy(channels[0].data, x_channel, channel_size);
  std::memcpy(channels[1].data, y_channel, channel_size);

  cv::merge(channels, 2, optical_flow);
}

void PwcNet::setImagesToInputLayer(const cv::Mat& source_image, const cv::Mat& dist_image) {
  std::vector<cv::Mat> channels;
  size_t channel_size = source_image.cols * source_image.rows;

  // Set source image
  cv::split(source_image, channels); // Split to BGR channels
  float *input_layer_blob = net_->blob_by_name(SOURCE_IMAGE_BLOB_)->mutable_cpu_data();
  // Store each channels to blob
  for (int i = 0; i < 3; i++)
    memcpy(input_layer_blob + (channel_size * i), channels[i].ptr<float>(), channel_size * sizeof(float));

  // Set dist image
  cv::split(dist_image, channels);
  input_layer_blob = net_->blob_by_name(DIST_IMAGE_BLOB_)->mutable_cpu_data();
  for (int i = 0; i < 3; i++)
    memcpy(input_layer_blob + (channel_size * i), channels[i].ptr<float>(), channel_size * sizeof(float));

  caffe::Caffe::set_mode(caffe::Caffe::GPU);
}

void PwcNet::visualizeOpticalFlow
(
  const cv::Mat& optical_flow,
  cv::Mat& visualized_optical_flow,
  float max_magnitude
) 
{
  cv::Mat hsv_image(optical_flow.rows, optical_flow.cols, CV_8UC3, cv::Vec3b(0, 0, 0));

  int total_pixels = optical_flow.total();
  for (int i = 0; i < total_pixels; i++)
  {
    const cv::Vec2f& flow_at_point = optical_flow.at<cv::Vec2f>(i);

    float flow_magnitude = 
      std::sqrt(flow_at_point[0]*flow_at_point[0] + flow_at_point[1]*flow_at_point[1]);
    float flow_direction = std::atan2(flow_at_point[0], flow_at_point[1]);

    uchar hue = (flow_direction / M_PI + 1.0) / 2.0 * 255;
    uchar saturation = std::min(std::max(flow_magnitude / max_magnitude, 0.0f), 1.0f) * 255;
    uchar value = 255;

    cv::Vec3b &hsv = hsv_image.at<cv::Vec3b>(i);
    hsv[0] = hue;
    hsv[1] = saturation;
    hsv[2] = value;
  }

  cv::cvtColor(hsv_image, visualized_optical_flow, cv::ColorConversionCodes::COLOR_HSV2BGR_FULL);
}

}
